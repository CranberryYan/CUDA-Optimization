// 内存对齐, 解决v3中的高利用率低吞吐率
// v1: Memory Throughput [%]	39.43
// v1: Memory Throughput [Gbyte/second]	289.31
// v1: L1/TEX Hit Rate [%]	16.67
// v2: Memory Throughput [%]	35.82
// v2: Memory Throughput [Gbyte/second]	262.15
// v3: Memory Throughput [%]	71.02
// v3: Memory Throughput [Gbyte/second]	281.60
// v3: L1/TEX Hit Rate [%]	77.25
// v3: L2 Hit Rate [%]	93.93

#include <hip/hip_runtime.h>

#include <time.h>
#include <algorithm>
#include <iostream>
#include <hip/hip_fp16.h>
using namespace std;
#define N 1 * 1024 * 1024
#define kBlockSize 256

// CUDA: grid stride looping
#define CUDA_1D_KERNEL_LOOP(i, n)                                                                 \
  for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x, step = blockDim.x * gridDim.x; i < (n); \
        i += step)

// Upsample Nearest2D Kernel is copyed from https://github.com/Oneflow-Inc/oneflow/blob/master/oneflow/user/kernels/upsample_nearest_kernel.cu#L78
template<typename T>
struct alignas(2 * sizeof(T)) Pack2X {
    T x;
    T y;
};

template<typename T>
__global__ void UpsampleNearest2D2XForward(const int32_t in_elem_cnt, const T* in_dptr,
                                           const int32_t in_height, const int32_t in_width,
                                           T* out_dptr) {
    const int32_t in_hw_size = in_width * in_height;
    CUDA_1D_KERNEL_LOOP(index, in_elem_cnt) {
        const T in_value = in_dptr[index];
        const int32_t nc_idx = index / in_hw_size;
        const int32_t hw_off = index - nc_idx * in_hw_size;
        const int32_t h = hw_off / in_width;
        const int32_t w = hw_off - h * in_width;
        Pack2X<T> out_value{in_value, in_value};
        Pack2X<T>* out_pack_dptr = reinterpret_cast<Pack2X<T>*>(out_dptr);
        out_pack_dptr[nc_idx * in_hw_size * 2 + h * 2 * in_width + w] = out_value;
        out_pack_dptr[nc_idx * in_hw_size * 2 + (h * 2 + 1) * in_width + w] = out_value;
    }
}

int main(){
    float *input_host = (float*)malloc(N*sizeof(float));
    float *input_device;
    hipMalloc((void **)&input_device, N*sizeof(float));
    for (int i = 0; i < N; i++) input_host[i] = 1.0;
    hipMemcpy(input_device, input_host, N*sizeof(float), hipMemcpyHostToDevice);

    float *output_host = (float*)malloc(N * 4 * sizeof(float));
    float *output_device;
    hipMalloc((void **)&output_device, N * 4 * sizeof(float));
    
    dim3 grid(N / kBlockSize, 1);
    dim3 block(kBlockSize, 1);
    UpsampleNearest2D2XForward<<<grid, block>>>(N, input_device, 1024, 1024, output_device);
    hipMemcpy(output_host, output_device, N * 4 * sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < 50; i++) {
        printf("%.5f\n", output_host[i]);
    }

    return 0;
}