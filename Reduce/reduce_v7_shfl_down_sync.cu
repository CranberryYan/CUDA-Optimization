// shfl_down_sync
// v0运行时间: 933.44us
// v0带宽利用率: 62.67%
// v0内存吞吐量: 144.33GB/s
// v1运行时间: 675.90us
// v1带宽利用率: 86.46%
// v1内存吞吐量: 199.35GB/s
// v2运行时间: 649.50us
// v2带宽利用率: 89.95%
// v2内存吞吐量: 207.42GB/s
// v3运行时间: 337.25us
// v3带宽利用率: 89.86%
// v3内存吞吐量: 398.79GB/s
// v4运行时间: 199.33us
// v4带宽利用率: 92.26%
// v4内存吞吐量: 674.85GB/s(比较接近760GB/s的理论值)
// v4: L1/TEX Cache Throughput [%]	68.56
// v4: L1/TEX Hit Rate [%]	0.27
// v5运行时间: 196.67us
// v5带宽利用率: 93.59%
// v5内存吞吐量: 683.72GB/s(比较接近760GB/s的理论值)
// v5: L1/TEX Cache Throughput [%]	69.61
// v5: L1/TEX Hit Rate [%]	0.35
// v6.0运行时间: 191.62us
// v6.0带宽利用率: 96.51%
// v6.0内存吞吐量: 700.47(比较接近760GB/s的理论值)
// v6.0: grid_size: [1024, 1, 1]
// v6.1运行时间: 191.04us
// v6.1带宽利用率: 96.32%
// v6.1内存吞吐量: 702.67(比较接近760GB/s的理论值)
// v6.1: grid_size: [2048, 1, 1]
// v6.2运行时间: 191.71us
// v6.2带宽利用率: 96.27%
// v6.2内存吞吐量: 700.13(比较接近760GB/s的理论值)
// v6.2: grid_size: [512, 1, 1]
// v7运行时间: 191.78us
// v7带宽利用率: 95.33%
// v7内存吞吐量: 699.88(比较接近760GB/s的理论值)

#include <hip/hip_runtime.h>

#include <time.h>
#include <vector>
#include <iostream>

#define N 32 * 1024 * 1024 // 32MB
#define BLOCK_SIZE 256
#define WARP_SIZE 32

void CPU_reduce(std::vector<float> &input_, double &output_) {
    for (auto x : input_) {
        output_ += x;
    }
}

bool checkout(float output_, float output_host) {
    if (std::abs(output_ - output_host) > 0.0001) {
        return false;
    } else {
        return true;
    }
}

template<unsigned int blockSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
    if(blockSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16);    // 0 + 16 
    if(blockSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);     // 0 + 8
    if(blockSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);      // 0 + 4
    if(blockSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);      // 0 + 2
    if(blockSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);      // 0 + 1
    return sum;
}

template<unsigned int blockSize, unsigned int NUM_PER_THREAD>
__global__ void reduce_v7(float *g_idata, float *g_odata) {
    float sum = 0;
    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * (blockDim.x * NUM_PER_THREAD) + threadIdx.x;

    // 一个thread要处理多个数据, 先加一起 -> 一个thread处理一个数据
    for (int i = 0; i < NUM_PER_THREAD; ++i) {
        sum += g_idata[gid + i * blockSize];
    }

    static __shared__ float warpLevelSums[WARP_SIZE];
    const int lane_id = threadIdx.x % WARP_SIZE; //warp中的tid
    const int warp_id = threadIdx.x / WARP_SIZE;

    sum = warpReduceSum<blockSize>(sum);

    if (lane_id == 0) {
        warpLevelSums[warp_id] = sum;
    }
    __syncthreads();

    sum = (tid < blockDim.x / WARP_SIZE) ? warpLevelSums[lane_id] : 0;
    if (warp_id == 0) {
        sum = warpReduceSum<blockSize/WARP_SIZE>(sum);
    }

    if (tid == 0) {
        // 写回每个block的sum
        g_odata[blockIdx.x] = sum;
    }
}

int main() {
    float *input_device;
    float *output_device;
    float *input_host = (float*)malloc(N * sizeof(float));
    float *output_host = (float*)malloc(N / BLOCK_SIZE * sizeof(float));
    hipMalloc((void**)&input_device, N * sizeof(float));
    hipMalloc((void**)&output_device, (N / BLOCK_SIZE) * sizeof(float));
    for (int i = 0; i < N; ++i) {
        input_host[i] = 1.0;
    }
    hipMemcpy(input_device, input_host, N * sizeof(float), hipMemcpyHostToDevice);

    const int block_num = 512;
    const int NUM_PER_BLOCK = N / block_num;                // 每个block处理多少数据
    const int NUM_PER_THREAD = NUM_PER_BLOCK / BLOCK_SIZE;  // 每个thread处理多少数据
    dim3 grid(block_num);
    dim3 block(BLOCK_SIZE);
    reduce_v7<BLOCK_SIZE, NUM_PER_THREAD><<<grid, block>>>(input_device, output_device);
    hipMemcpy(output_host, output_device, block_num * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 1; i < block_num; ++i) {
        output_host[0] += output_host[i];
    }

    bool res;
    std::vector<float> input_(N, 1.0);
    double output_ = 0;
    CPU_reduce(input_, output_);
    res = checkout(output_, output_host[0]);
    if (res) {
        std::cout << "PASSED!" << std::endl;
        std::cout << "CPU: " << output_ << std::endl;
        std::cout << "GPU: " << output_host[0] << std::endl;
    } else {
        std::cout << "FAILED!" << std::endl;
        std::cout << "CPU: " << output_ << std::endl;
        std::cout << "GPU: " << output_host[0] << std::endl;
    }

    free(input_host);
    free(output_host);
    hipFree(input_device);
    hipFree(output_device);

    return 0;
}