// v0:
//  1.49ms
//	Compute (SM) Throughput [%]	92.10
//	Memory Throughput [Gbyte/second]	8.43
// v1:
//	使用shared_memory
//	性能分析意义不大, 为了强行适配该版本的改动
//	shape做出调整
//	Memory Throughput [Gbyte/second]	12.33
// v2:
//  使用shared_memory + 分块计算
//  Memory Throughput [Gbyte/second]	11.34
// v2_2:
//	重新分配BLOCK_SIZE, 适应shared_memory和block中最大thread限制
//	Memory Throughput [Gbyte/second]	12.31
// v2_3:
//  无性能优化, 根据自己的理解更改索引方式
// v3:
//  增加每个thread的工作量, 减少block的数量
//  297.82us
//  Memory Throughput [Gbyte/second]	14.77
// v4:
//	向量化
//	Memory Throughput [Gbyte/second]	37.36
// v5:
//	内积 -> 外积
// 	Memory Throughput [Gbyte/second]	54.32

#include <hip/hip_runtime.h>
#include <iostream>
#define STRIDE 2
#define BLOCK_SIZE 16
#define NUM_PER_REG  		2
#define NUM_PER_THREAD  4
#define M_NUM_PER_BLOCK 32
#define N_NUM_PER_BLOCK 32
#define K_NUM_PER_BLOCK 32
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

void random_matrix(int m, int n, float *mat) {
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			mat[i * n + j] = 2.0 * static_cast<float>(drand48()) - 1.0;
		}
	}
}

// A: [M, K]   B: [K, N]
void sgemm_CPU(float *A, float *B, float *C,
	const int M, const int N, const int K) {
	for (int m = 0; m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			float temp = 0.0f;
			for (int k = 0; k < K; ++k) {
				temp += A[m*K + k] * B[k*N + n];
			}
			C[m*N + n] = temp;
			// printf("C[%d]: %f", m*N + n, temp);
		}
	}
}

bool checkout(float *C_buf_host_cpu, float *C_buf_host_gpu,
	const int M, const int N) {
	for (int m = 0;  m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			if (std::abs(C_buf_host_cpu[m*N+n] -
				C_buf_host_gpu[m*N+n]) > 1e-3) {
				std::cout << "FAILED!" << std::endl;
				std::cout << "C_buf_host_cpu[" << m*N+n << "]: "
					<< C_buf_host_cpu[m*N+n] << std::endl
					<< "C_buf_host_gpu[" << m*N+n << "]: "
					<< C_buf_host_gpu[m*N+n] << std::endl;
				return false;
			}
		}
	}
	std::cout << "PASSED!" << std::endl;
	return true;
}

template<unsigned int M_NUM_PER_BLOCK_, unsigned int N_NUM_PER_BLOCK_,
  unsigned int K_NUM_PER_BLOCK_, unsigned int NUM_PER_THREAD_,
	unsigned int NUM_PER_REG_>
__global__ void sgemm_v5(float *A, float *B, float *C,
	const int M, const int N, const int K) {
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int ctx = tid % 16;
	int cty = tid / 16;
	// A矩阵的小块先偏移到所在的行
  float *A_ptr = A + (blockIdx.y * M_NUM_PER_BLOCK_) * K;
	// B矩阵的小块先偏移到所在的列
  float *B_ptr = B + blockIdx.x * N_NUM_PER_BLOCK_;
	// C矩阵偏移到所在的行 + 列
	float *C_ptr = C + (blockIdx.y * M_NUM_PER_BLOCK_) * K +
		blockIdx.x * N_NUM_PER_BLOCK_;

	float a_reg[NUM_PER_REG_] = {0.0f};
	float b_reg[NUM_PER_REG_] = {0.0f};
  float temp[NUM_PER_REG_][NUM_PER_REG_] = {0.0f};
  __shared__ float a_shared[M_NUM_PER_BLOCK_][K_NUM_PER_BLOCK_];
  __shared__ float b_shared[K_NUM_PER_BLOCK_][N_NUM_PER_BLOCK_];

  for (int k_ = 0; k_ < K; k_ += K_NUM_PER_BLOCK_) {
		// A_ptr:
		//	threadIdx.y * K: 偏移到当前的小块所在的行的起始地址
		//	k_: 偏移到当前的小块所在的列的起始地址(每个小块处理K_NUM_PER_BLOCK_个元素)
		// B_ptr: 
		//	(threadIdx.y + k) * N: 偏移到当前的小块所在的行的起始地址
		FETCH_FLOAT4(a_shared[threadIdx.y][threadIdx.x * NUM_PER_THREAD_]) =
			FETCH_FLOAT4(A_ptr[threadIdx.y * K + k_ + threadIdx.x * NUM_PER_THREAD_]);
		FETCH_FLOAT4(b_shared[threadIdx.y][threadIdx.x * NUM_PER_THREAD_]) =
			FETCH_FLOAT4(B_ptr[(threadIdx.y + k_) * N + threadIdx.x * NUM_PER_THREAD_]);
		__syncthreads();

		for (int k = 0; k < K_NUM_PER_BLOCK_; ++k) {
			// 一个thread处理4个元素, a/b各取2个元素
			// 且重新分配ty/tx
			a_reg[0] = a_shared[cty * NUM_PER_REG_][k];
			a_reg[1] = a_shared[cty * NUM_PER_REG_ + 1][k];
			b_reg[0] = b_shared[k][ctx * NUM_PER_REG_];
			b_reg[1] = b_shared[k][ctx * NUM_PER_REG_ + 1];
			for (int m = 0; m < NUM_PER_REG_; ++m) {
				for (int n = 0; n < NUM_PER_REG_; ++n) {
					temp[m][n] += a_reg[m] * b_reg[n];
				}
			}
		}
		__syncthreads();
	}

	for (int m = 0; m < NUM_PER_REG_; ++m) {
		for (int n = 0; n < NUM_PER_REG_; ++n) {
			C_ptr[(cty * NUM_PER_REG_ + m) * N + ctx * NUM_PER_REG_ + n] =
				temp[m][n];
		}
	}
}

int main() {
	// lhs: [M, K]
	// rhs: [K, N]
	const unsigned int m = 1024;
	const unsigned int n = 1024;
	const unsigned int k = 1024;

	// host
	std::cout << " ============= host ============= " << std::endl;
	float *A_buf_host = (float*)malloc(m * k *sizeof(float));
	float *B_buf_host = (float*)malloc(k * n *sizeof(float));
	float *C_buf_host_cpu = (float*)malloc(m * n *sizeof(float));
	float *C_buf_host_gpu = (float*)malloc(m * n *sizeof(float));
	random_matrix(m, k, A_buf_host);
	random_matrix(k, n, B_buf_host);
	memset(C_buf_host_cpu, 0, m * n *sizeof(float));
	memset(C_buf_host_gpu, 0, m * n *sizeof(float));

	// device
	std::cout << " ============= device ============= " << std::endl;
	float *A_buf_device, *B_buf_device, *C_buf_device;
	hipMalloc((void**)&A_buf_device, m * k *sizeof(float));
	hipMalloc((void**)&B_buf_device, m * k *sizeof(float));
	hipMalloc((void**)&C_buf_device, m * k *sizeof(float));
	hipMemcpy(A_buf_device, A_buf_host, m * k *sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(B_buf_device, B_buf_host, k * n *sizeof(float),
		hipMemcpyHostToDevice);

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipFuncCache_t cacheConfig;
	hipDeviceProp_t deviceProp;
	hipDeviceGetCacheConfig(&cacheConfig);
	hipGetDeviceProperties(&deviceProp, 0);
	switch (cacheConfig) {
			case hipFuncCachePreferNone:
					std::cout << "Current cache config: PreferNone" << std::endl;
					break;
			case hipFuncCachePreferShared:
					std::cout << "Current cache config: PreferShared" << std::endl;
					break;
			case hipFuncCachePreferL1:
					std::cout << "Current cache config: PreferL1" << std::endl;
					break;
			case hipFuncCachePreferEqual:
					std::cout << "Current cache config: PreferEqual" << std::endl;
					break;
	}
	std::cout << "Max Shared Memory per Block: "
		<< deviceProp.sharedMemPerBlock << " bytes" << std::endl;
	std::cout << "Max Shared Memory per SM: "
		<< deviceProp.sharedMemPerMultiprocessor << " bytes" << std::endl;

	// CPU_segmm
	std::cout << " ============= CPU_segmm ============= " << std::endl;
	sgemm_CPU(A_buf_host, B_buf_host, C_buf_host_cpu,
		m, n, k);

	// GPU_segmm
  // before:
  //  增加每个thread的工作量
  //    -> 减少block的数量
  //  girdDim: [64, 64] -> [32, 32]
  //  blockDim: [16, 16] -> 一个block有256个thread, 一个thread处理4个元素
  // after:
  //  四个为一组进行处理, 1024/32/4 -> 8 -> blockDim.x = 8
  //  blockDim.y = 1024/32 - > 32
	std::cout << " ============= GPU_segmm ============= " << std::endl;
	dim3 grid((n + N_NUM_PER_BLOCK - 1) / N_NUM_PER_BLOCK,
    (m + M_NUM_PER_BLOCK - 1) / M_NUM_PER_BLOCK);
	dim3 block(8, 32);
	sgemm_v5<M_NUM_PER_BLOCK, N_NUM_PER_BLOCK, K_NUM_PER_BLOCK,
		NUM_PER_THREAD, NUM_PER_REG>
    <<<grid, block>>>( A_buf_device, B_buf_device, C_buf_device, m, n, k);

	// verify
	std::cout << " ============= VERIFY ============= " << std::endl;
	hipMemcpy(C_buf_host_gpu, C_buf_device, m * k *sizeof(float),
		hipMemcpyDeviceToHost);
	bool res = checkout(C_buf_host_cpu, C_buf_host_gpu,
		m, n);

	// free
	free(A_buf_host);
	free(B_buf_host);
	free(C_buf_host_cpu);
	free(C_buf_host_gpu);
	hipFree(A_buf_device);
	hipFree(B_buf_device);
	hipFree(C_buf_device);

	return 0;
}