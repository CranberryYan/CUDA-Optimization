
#include <hip/hip_runtime.h>
#include <iostream>
#define BLOCK_SIZE 16

void random_matrix(int m, int n, float *mat) {
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			mat[i * n + j] = 2.0 * static_cast<float>(drand48()) - 1.0;
		}
	}
}

// A: [M, K]   B: [K, N]
void sgemm_CPU(float *A, float *B, float *C,
	const int M, const int N, const int K) {
	for (int m = 0; m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			float temp = 0.0f;
			for (int k = 0; k < K; ++k) {
				temp += A[m*K + k] * B[k*N + n];
			}
			// printf("C[%d]: %f", m*N + n, temp);
			C[m*N + n] = temp;
		}
	}
}

bool checkout(float *C_buf_host_cpu, float *C_buf_host_gpu,
	const int M, const int N) {
	for (int m = 0;  m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			if (C_buf_host_cpu[m*N+n] != C_buf_host_gpu[m*N+n]) {
				std::cout << "FAILED!" << std::endl;
				std::cout << "C_buf_host_cpu[" << m*N+n << "]: " << C_buf_host_cpu[m*N+n] << std::endl
					<< "C_buf_host_gpu[" << m*N+n << "]: " << C_buf_host_gpu[m*N+n] << std::endl;
				return false;
			}
		}
	}

	std::cout << "PASSED!" << std::endl;
	return true;
}

__global__ void sgemm_v0(float *A, float *B, float *C,
	const int M, const int N, const int K) {

}

int main() {
	printf("gemm_baseline\n");
	
	// lhs: [M, K]
	// rhs: [K, N]
	const int m = 1024;
	const int n = 1024;
	const int k = 1024;

	// host
	std::cout << " ============= host ============= " << std::endl;
	float *A_buf_host = (float*)malloc(m * k *sizeof(float));
	float *B_buf_host = (float*)malloc(k * n *sizeof(float));
	float *C_buf_host_cpu = (float*)malloc(m * n *sizeof(float));
	float *C_buf_host_gpu = (float*)malloc(m * n *sizeof(float));
	random_matrix(m, k, A_buf_host);
	random_matrix(k, n, B_buf_host);
	memset(C_buf_host_cpu, 0, m * n *sizeof(float));
	memset(C_buf_host_gpu, 0, m * n *sizeof(float));

	// device
	std::cout << " ============= device ============= " << std::endl;
	float *A_buf_device, *B_buf_device, *C_buf_device;
	hipMalloc((void**)&A_buf_device, m * k *sizeof(float));
	hipMalloc((void**)&B_buf_device, m * k *sizeof(float));
	hipMalloc((void**)&C_buf_device, m * k *sizeof(float));
	hipMemcpy(A_buf_device, A_buf_host, m * k *sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(B_buf_device, B_buf_host, k * n *sizeof(float),
		hipMemcpyHostToDevice);

	// CPU_segmm
	std::cout << " ============= CPU_segmm ============= " << std::endl;
	sgemm_CPU(A_buf_host, B_buf_host, C_buf_host_cpu,
		m, n, k);

	// GPU_segmm
	std::cout << " ============= GPU_segmm ============= " << std::endl;
	dim3 grid((m + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	sgemm_v0<<<grid, block>>>(A_buf_device, B_buf_device, C_buf_device,
		m, n, k);

	// verify
	std::cout << " ============= VERIFY ============= " << std::endl;
	hipMemcpy(C_buf_host_gpu, C_buf_device, m * k *sizeof(float),
		hipMemcpyDeviceToHost);
	bool res = checkout(C_buf_host_cpu, C_buf_host_gpu,
		m, n);

	// free
	free(A_buf_host);
	free(B_buf_host);
	free(C_buf_host_cpu);
	free(C_buf_host_gpu);
	hipFree(A_buf_device);
	hipFree(B_buf_device);
	hipFree(C_buf_device);

	return 0;
}