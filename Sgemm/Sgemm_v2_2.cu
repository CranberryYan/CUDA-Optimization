// v0:
//	Compute (SM) Throughput [%]	92.10
//	Memory Throughput [Gbyte/second]	8.43
// v1:
//	使用shared_memory
//	性能分析意义不大, 为了强行适配该版本的改动
//	shape做出调整
//	Memory Throughput [Gbyte/second]	12.33
// v2:
//  使用shared_memory + 分块计算
//  Memory Throughput [Gbyte/second]	11.34
// v2_2:
//	重新分配BLOCK_SIZE, 适应shared_memory和block中最大thread限制
//	Memory Throughput [Gbyte/second]	12.31

#include <hip/hip_runtime.h>
#include <iostream>
#define BLOCK_SIZE 32

void random_matrix(int m, int n, float *mat) {
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			mat[i * n + j] = 2.0 * static_cast<float>(drand48()) - 1.0;
		}
	}
}

// A: [M, K]   B: [K, N]
void sgemm_CPU(float *A, float *B, float *C,
	const int M, const int N, const int K) {
	for (int m = 0; m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			float temp = 0.0f;
			for (int k = 0; k < K; ++k) {
				temp += A[m*K + k] * B[k*N + n];
			}
			C[m*N + n] = temp;
			// printf("C[%d]: %f", m*N + n, temp);
		}
	}
}

bool checkout(float *C_buf_host_cpu, float *C_buf_host_gpu,
	const int M, const int N) {
	for (int m = 0;  m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			if (std::abs(C_buf_host_cpu[m*N+n] -
				C_buf_host_gpu[m*N+n]) > 1e-3) {
				std::cout << "FAILED!" << std::endl;
				std::cout << "C_buf_host_cpu[" << m*N+n << "]: "
					<< C_buf_host_cpu[m*N+n] << std::endl
					<< "C_buf_host_gpu[" << m*N+n << "]: "
					<< C_buf_host_gpu[m*N+n] << std::endl;
				return false;
			}
		}
	}

	std::cout << "PASSED!" << std::endl;
	return true;
}

// BLOCK_SIZE: 32
// 每个block有32个thread, 每个thread负责一个元素
// 	每个block负责32*32个元素, 一共32*32个block
//	-> 1024*1024个元素
template<unsigned int BLOCK_DIM>
__global__ void sgemm_v2_2(float *A, float *B, float *C,
	const int M, const int N, const int K) {
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	// block_level
	// blockDim: [32, 32]
	// blockIdx: [0, 31]
	//	offset_stride: 32
	// 行的偏移(y轴)
	// A: [M, K]
	//	M: 一行K个元素, 一共M行, 一个block负责blockDim.y(32)行,
	//		一共gridDim.y(32)个block
	int offset_row = blockIdx.y * blockDim.y * K;
	// 列的偏移(x轴)
	// B: [K, N]
	//	N: 一行N个元素, 每个block负责blockDim.x(32)列,
	//		一共gridDim.x(32)个block
	int offset_col = blockIdx.x * blockDim.x;
	float *A_ptr = A + offset_row;
	float *B_ptr = B + offset_col;

	// 注: smem的大小需要在编译期确定
	//	-> template
	// 3080: L1 cache + smem = 8704KB
	//	单个SM: 100KB(max)
	//	单个block: 48KB(max)
	//		-> 48 * 1024 / 4(FP32) = 12 * 1024个FP32
	//		-> 12 * 1024 / 2 = 6 * 1024个FP32
	//		-> sqrt(6 * 1024) -> 2 * 32 -> 64
	//		-> block中最多有1024个thread -> 32
	float temp = 0.0f;
	__shared__ float a_shared[BLOCK_DIM][BLOCK_DIM];
	__shared__ float b_shared[BLOCK_DIM][BLOCK_DIM];
	for (int k = 0; k < K; k += blockDim.x) {
		a_shared[threadIdx.y][threadIdx.x] =
			A_ptr[threadIdx.y * K + threadIdx.x + k];
		b_shared[threadIdx.y][threadIdx.x] =
			B_ptr[(threadIdx.y + k) * N + threadIdx.x];
    __syncthreads();
    for (int i = 0; i < BLOCK_DIM; ++i) {
      temp += a_shared[threadIdx.y][i] * b_shared[i][threadIdx.x];
    }
    __syncthreads();
	}

  C[y * N + x] = temp;
}

int main() {
	// lhs: [M, K]
	// rhs: [K, N]
	const unsigned int m = 1024;
	const unsigned int n = 1024;
	const unsigned int k = 1024;

	// host
	std::cout << " ============= host ============= " << std::endl;
	float *A_buf_host = (float*)malloc(m * k *sizeof(float));
	float *B_buf_host = (float*)malloc(k * n *sizeof(float));
	float *C_buf_host_cpu = (float*)malloc(m * n *sizeof(float));
	float *C_buf_host_gpu = (float*)malloc(m * n *sizeof(float));
	random_matrix(m, k, A_buf_host);
	random_matrix(k, n, B_buf_host);
	memset(C_buf_host_cpu, 0, m * n *sizeof(float));
	memset(C_buf_host_gpu, 0, m * n *sizeof(float));

	// device
	std::cout << " ============= device ============= " << std::endl;
	float *A_buf_device, *B_buf_device, *C_buf_device;
	hipMalloc((void**)&A_buf_device, m * k *sizeof(float));
	hipMalloc((void**)&B_buf_device, m * k *sizeof(float));
	hipMalloc((void**)&C_buf_device, m * k *sizeof(float));
	hipMemcpy(A_buf_device, A_buf_host, m * k *sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(B_buf_device, B_buf_host, k * n *sizeof(float),
		hipMemcpyHostToDevice);

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipFuncCache_t cacheConfig;
	hipDeviceProp_t deviceProp;
	hipDeviceGetCacheConfig(&cacheConfig);
	hipGetDeviceProperties(&deviceProp, 0);
	switch (cacheConfig) {
			case hipFuncCachePreferNone:
					std::cout << "Current cache config: PreferNone" << std::endl;
					break;
			case hipFuncCachePreferShared:
					std::cout << "Current cache config: PreferShared" << std::endl;
					break;
			case hipFuncCachePreferL1:
					std::cout << "Current cache config: PreferL1" << std::endl;
					break;
			case hipFuncCachePreferEqual:
					std::cout << "Current cache config: PreferEqual" << std::endl;
					break;
	}
	std::cout << "Max Shared Memory per Block: "
		<< deviceProp.sharedMemPerBlock << " bytes" << std::endl;
	std::cout << "Max Shared Memory per SM: "
		<< deviceProp.sharedMemPerMultiprocessor << " bytes" << std::endl;

	// CPU_segmm
	std::cout << " ============= CPU_segmm ============= " << std::endl;
	sgemm_CPU(A_buf_host, B_buf_host, C_buf_host_cpu,
		m, n, k);

	// GPU_segmm
	// sgemm: 二维 -> grid和block都是二维
	std::cout << " ============= GPU_segmm ============= " << std::endl;
	dim3 grid((m + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(m + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	sgemm_v2_2<BLOCK_SIZE><<<grid, block>>>(
		A_buf_device, B_buf_device, C_buf_device,
		m, n, k);

	// verify
	std::cout << " ============= VERIFY ============= " << std::endl;
	hipMemcpy(C_buf_host_gpu, C_buf_device, m * k *sizeof(float),
		hipMemcpyDeviceToHost);
	bool res = checkout(C_buf_host_cpu, C_buf_host_gpu,
		m, n);

	// free
	free(A_buf_host);
	free(B_buf_host);
	free(C_buf_host_cpu);
	free(C_buf_host_gpu);
	hipFree(A_buf_device);
	hipFree(B_buf_device);
	hipFree(C_buf_device);

	return 0;
}