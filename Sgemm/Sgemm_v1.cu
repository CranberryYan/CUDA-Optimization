// v0: 
//	Compute (SM) Throughput [%]	92.10
//	Memory Throughput [Gbyte/second]	8.43
// v1: 
//	使用shared_memory
//	性能分析意义不大, 为了强行适配该版本的改动
//	shape做出调整
//	Memory Throughput [Gbyte/second]	12.33

#include <hip/hip_runtime.h>
#include <iostream>
#define BLOCK_SIZE 16

void random_matrix(int m, int n, float *mat) {
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			mat[i * n + j] = 2.0 * static_cast<float>(drand48()) - 1.0;
		}
	}
}

// A: [M, K]   B: [K, N]
void sgemm_CPU(float *A, float *B, float *C,
	const int M, const int N, const int K) {
	for (int m = 0; m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			float temp = 0.0f;
			for (int k = 0; k < K; ++k) {
				temp += A[m*K + k] * B[k*N + n];
			}
			C[m*N + n] = temp;
			// printf("C[%d]: %f", m*N + n, temp);
		}
	}
}

bool checkout(float *C_buf_host_cpu, float *C_buf_host_gpu,
	const int M, const int N) {
	for (int m = 0;  m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			if (std::abs(C_buf_host_cpu[m*N+n] -
				C_buf_host_gpu[m*N+n]) > 1e-3) {
				std::cout << "FALED!" << std::endl;
				std::cout << "C_buf_host_cpu[" << m*N+n << "]: " << C_buf_host_cpu[m*N+n] << std::endl
					<< "C_buf_host_gpu[" << m*N+n << "]: " << C_buf_host_gpu[m*N+n] << std::endl;
				return false;
			}
		}
	}

	std::cout << "PASSED!" << std::endl;
	return true;
}

// BLOCK_SIZE: 16
// 每个block有16个thread, 每个thread负责一个元素
// 	每个block负责16*16个元素, 一共64*64个block
//	-> 1024*1024个元素
template<unsigned int BLOCK_DIM, unsigned int K_>
__global__ void sgemm_v1(float *A, float *B, float *C,
	const int M, const int N, const int K) {
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	// block_level
	// blockDim: [16, 16]
	// blockIdx: [0, 63]
	//	offset_stride: 16
	// 行的偏移(y轴)
	int offset_row = blockIdx.y * blockDim.y * K;
	// 列的偏移(x轴)
	int offset_col = blockIdx.x * blockDim.x;
	float *A_ptr = A + offset_row;
	float *B_ptr = B + offset_col;

	// 注: smem的大小需要在编译期确定
	//	-> template
	// 3080: L1 cache + smem = 8704KB
	//	单个SM: 100KB(max) 
	//	单个block: 48KB(max)
	//		-> 48 * 1024 / 4(FP32) / BLOCK_DIM / 2 -> K_: 384个FP32
	__shared__ float a_shared[BLOCK_DIM][K_];
	__shared__ float b_shared[K_][BLOCK_DIM];
	for (int k = 0; k < K; k += blockDim.x) {
		a_shared[threadIdx.y][threadIdx.x + k] =
			A_ptr[threadIdx.y * K + threadIdx.x + k];
		b_shared[threadIdx.y + k][threadIdx.x] =
			B_ptr[(threadIdx.y + k) * N + threadIdx.x];
	}
	__syncthreads();

	// thread_level
	float temp = 0.0f;
	for (int k = 0; k < K; ++k) {
		temp += a_shared[threadIdx.y][k] *
			b_shared[k][threadIdx.x];
	}
	int offset_C = y * N + x;
	C[offset_C] = temp;
}

int main() {
	// lhs: [M, K]
	// rhs: [K, N]
	printf("gemm_baseline\n");
	const unsigned int m = 384;
	const unsigned int n = 384;
	const unsigned int k = 384;

	// host
	std::cout << " ============= host ============= " << std::endl;
	float *A_buf_host = (float*)malloc(m * k *sizeof(float));
	float *B_buf_host = (float*)malloc(k * n *sizeof(float));
	float *C_buf_host_cpu = (float*)malloc(m * n *sizeof(float));
	float *C_buf_host_gpu = (float*)malloc(m * n *sizeof(float));
	random_matrix(m, k, A_buf_host);
	random_matrix(k, n, B_buf_host);
	memset(C_buf_host_cpu, 0, m * n *sizeof(float));
	memset(C_buf_host_gpu, 0, m * n *sizeof(float));

	// device
	std::cout << " ============= device ============= " << std::endl;
	float *A_buf_device, *B_buf_device, *C_buf_device;
	hipMalloc((void**)&A_buf_device, m * k *sizeof(float));
	hipMalloc((void**)&B_buf_device, m * k *sizeof(float));
	hipMalloc((void**)&C_buf_device, m * k *sizeof(float));
	hipMemcpy(A_buf_device, A_buf_host, m * k *sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(B_buf_device, B_buf_host, k * n *sizeof(float),
		hipMemcpyHostToDevice);

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipFuncCache_t cacheConfig;
	hipDeviceProp_t deviceProp;
	hipDeviceGetCacheConfig(&cacheConfig);
	hipGetDeviceProperties(&deviceProp, 0);
	switch (cacheConfig) {
			case hipFuncCachePreferNone:
					std::cout << "Current cache config: PreferNone" << std::endl;
					break;
			case hipFuncCachePreferShared:
					std::cout << "Current cache config: PreferShared" << std::endl;
					break;
			case hipFuncCachePreferL1:
					std::cout << "Current cache config: PreferL1" << std::endl;
					break;
			case hipFuncCachePreferEqual:
					std::cout << "Current cache config: PreferEqual" << std::endl;
					break;
	}
	std::cout << "Max Shared Memory per Block: " << deviceProp.sharedMemPerBlock << " bytes" << std::endl;
	std::cout << "Max Shared Memory per SM: " << deviceProp.sharedMemPerMultiprocessor << " bytes" << std::endl;

	// CPU_segmm
	std::cout << " ============= CPU_segmm ============= " << std::endl;
	sgemm_CPU(A_buf_host, B_buf_host, C_buf_host_cpu,
		m, n, k);

	// GPU_segmm
	// sgemm: 二维 -> grid和block都是二维
	std::cout << " ============= GPU_segmm ============= " << std::endl;
	dim3 grid((m + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	sgemm_v1<BLOCK_SIZE, k><<<grid, block>>>(
		A_buf_device, B_buf_device, C_buf_device,
		m, n, k);

	// verify
	std::cout << " ============= VERIFY ============= " << std::endl;
	hipMemcpy(C_buf_host_gpu, C_buf_device, m * k *sizeof(float),
		hipMemcpyDeviceToHost);
	bool res = checkout(C_buf_host_cpu, C_buf_host_gpu,
		m, n);

	// free
	free(A_buf_host);
	free(B_buf_host);
	free(C_buf_host_cpu);
	free(C_buf_host_gpu);
	hipFree(A_buf_device);
	hipFree(B_buf_device);
	hipFree(C_buf_device);

	return 0;
}